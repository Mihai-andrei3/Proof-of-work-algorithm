#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

__const uint64_t limit = 1e8; //MAX NONCE
__device__ int global_flag = 0; //Flag to mark if a thread has found the nonce

//function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce(
    BYTE *block_content, // the content of the block
    BYTE *block_hash, // the hash for the block to be returned
    BYTE *DIFFICULTY, // difficulty
    uint64_t *result_nonce //result nonce to be returned
) {
	//Compute the current length of the block content
    uint64_t current_length = d_strlen((char*)block_content);
	//Compute the nonce corresponding to this thread
    uint64_t nonce = blockIdx.x * blockDim.x + threadIdx.x + 1;
	//Temporary variables to eliminate concurrency issues
    char nonce_string[NONCE_SIZE]; // to store the nonce as a string
	BYTE temp_block_hash[SHA256_HASH_SIZE]; 
	BYTE temp_block_content[BLOCK_SIZE]; 

	//Check if the nonce has been found by another thread
	if (global_flag != 0) {
		return;
	}

	//Copy the block content to the temporary variable
	d_strcpy((char*)temp_block_content, (const char*)block_content);

	//Check if the nonce is within the limit
    if (nonce <= limit) {
		//Convert the nonce to a string
        intToString(nonce, nonce_string);
		//Append the nonce to the block content
        d_strcpy((char*)temp_block_content + current_length, nonce_string);
		//Compute the hash
        apply_sha256(temp_block_content, d_strlen((const char*)temp_block_content) , temp_block_hash, 1); 

		//Check if the hash is less than the difficulty and no other thread has found the nonce
        if (compare_hashes(temp_block_hash, DIFFICULTY) <= 0 && global_flag == 0) {
			//Update the result nonce and block hash
			*result_nonce = nonce;
			d_strcpy((char*)block_hash, (const char*)temp_block_hash);
			//Set the flag to 1
			atomicAdd(&global_flag, 1);
			return;
        }

    }

}




int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; 
	uint64_t nonce = 0; // The nonce to be found

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);

	hipEvent_t start, stop;
	startTiming(&start, &stop);

	//Commpute the number of blocks
	const size_t block_size = 256; 
    size_t blocks_no = MAX_NONCE / block_size;
 
	//add an extra block if the division is not exact
    if ((int)MAX_NONCE % block_size != 0) 
    	++blocks_no;

	//device variables
    BYTE *device_block_content = 0;
    BYTE *device_block_hash = 0;
    BYTE *DIFF = 0;
	uint64_t *result_nonce = 0;

	//Allocate memory for the device variables
	hipMalloc(&result_nonce, sizeof(uint64_t));
    hipMalloc(&device_block_hash, SHA256_HASH_SIZE);
    hipMalloc(&device_block_content, BLOCK_SIZE);
	hipMalloc(&DIFF, SHA256_HASH_SIZE);
	//Copy the data from the host to the device
	hipMemcpy(device_block_content, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(device_block_hash, block_hash, SHA256_HASH_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(DIFF, DIFFICULTY, SHA256_HASH_SIZE, hipMemcpyHostToDevice);

    //Check if the memory was allocated successfully
    if (device_block_content == 0 || device_block_hash == 0  || DIFF == 0) {
		printf("[DEVICE] Couldn't allocate memory\n");
   		return 1;
  	}

	//Call the kernel function
	findNonce<<<blocks_no, block_size>>>(device_block_content, device_block_hash, DIFF, result_nonce);
	hipDeviceSynchronize();

	//Copy the results from the device to the host
	hipMemcpy(&nonce, result_nonce, sizeof(uint64_t), hipMemcpyDeviceToHost);
	hipMemcpy(block_hash, device_block_hash, SHA256_HASH_SIZE, hipMemcpyDeviceToHost);

	//Free the memory
	hipFree(device_block_content);
	hipFree(device_block_hash);
	hipFree(DIFF);
	hipFree(result_nonce);

	//Print the result
	float seconds = stopTiming(&start, &stop);
	printResult(block_hash, nonce, seconds);

	return 0;
}
